#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "drillx.h"
#include "equix.h"
#include "hashx.h"
#include "equix/src/context.h"
#include "equix/src/solver.h"
#include "equix/src/solver_heap.h"
#include "hashx/src/context.h"

const int BATCH_SIZE = 512;

extern "C" void hash(uint8_t *challenge, uint8_t *nonce, uint64_t *out) {
    // Allocate pinned memory for ctxs and hash_space
    hashx_ctx** ctxs;
    uint64_t** hash_space;
    if (hipHostAlloc(&ctxs, BATCH_SIZE * sizeof(hashx_ctx*), hipHostMallocDefault) != hipSuccess) {
        printf("Failed to allocate pinned memory for ctxs\n");
        return;
    }
    if (hipHostAlloc(&hash_space, BATCH_SIZE * sizeof(uint64_t*), hipHostMallocDefault) != hipSuccess) {
        printf("Failed to allocate pinned memory for hash_space\n");
        hipHostFree(ctxs);
        return;
    }
    for (int i = 0; i < BATCH_SIZE; i++) {
        if (hipMalloc(&hash_space[i], INDEX_SPACE * sizeof(uint64_t)) != hipSuccess) {
            printf("Failed to allocate device memory for hash_space[%d]\n", i);
            hipHostFree(hash_space);
            hipHostFree(ctxs);
            return;
        }
    }

    // Prepare seed and hash contexts
    uint8_t seed[40];
    memcpy(seed, challenge, 32);
    for (int i = 0; i < BATCH_SIZE; i++) {
        uint64_t nonce_offset = *((uint64_t*)nonce) + i;
        memcpy(seed + 32, &nonce_offset, 8);
        ctxs[i] = hashx_alloc(HASHX_INTERPRETED);
        if (!ctxs[i] || !hashx_make(ctxs[i], seed, 40)) {
            printf("Failed to make hash\n");
            hipHostFree(hash_space);
            hipHostFree(ctxs);
            return;
        }
    }

    // Launch kernel to parallelize hashx operations
    dim3 threadsPerBlock(256); // 256 threads per block
    dim3 blocksPerGrid((65536 * BATCH_SIZE + threadsPerBlock.x - 1) / threadsPerBlock.x); // enough blocks to cover batch
    do_hash_stage0i<<<blocksPerGrid, threadsPerBlock>>>(ctxs, hash_space);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel launch or synchronization error: %s\n", hipGetErrorString(err));
        hipHostFree(hash_space);
        hipHostFree(ctxs);
        return;
    }

    // Copy hashes back to cpu
    for (int i = 0; i < BATCH_SIZE; i++) {
        if (hipMemcpy(out + i * INDEX_SPACE, hash_space[i], INDEX_SPACE * sizeof(uint64_t), hipMemcpyDeviceToHost) != hipSuccess) {
            printf("Failed to copy hash_space[%d] to host\n", i);
            hipHostFree(hash_space);
            hipHostFree(ctxs);
            return;
        }
    }

    // Free memory
    for (int i = 0; i < BATCH_SIZE; i++) {
        hashx_free(ctxs[i]);
        hipFree(hash_space[i]);
    }
    hipHostFree(hash_space);
    hipHostFree(ctxs);

    // Print errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}

__global__ void do_hash_stage0i(hashx_ctx** ctxs, uint64_t** hash_space) {
    uint32_t item = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t batch_idx = item / INDEX_SPACE;
    uint32_t i = item % INDEX_SPACE;
    if (batch_idx < BATCH_SIZE) {
        hash_stage0i(ctxs[batch_idx], hash_space[batch_idx], i);
    }
}

extern "C" void solve_all_stages(uint64_t *hashes, uint8_t *out, uint32_t *sols) {
    // Create an equix context
    equix_ctx* ctx = equix_alloc(EQUIX_CTX_SOLVE);
    if (ctx == nullptr) {
        printf("Failed to allocate equix context\n");
        return;
    }

    // Do the remaining stages
    equix_solution solutions[EQUIX_MAX_SOLS];
    uint32_t num_sols = equix_solver_solve(hashes, ctx->heap, solutions);

    // Copy results back to host
    memcpy(sols, &num_sols, sizeof(num_sols));
    if (num_sols > 0) {
        memcpy(out, solutions[0].idx, sizeof(solutions[0].idx));
    }

    // Free memory
    equix_free(ctx);
}
